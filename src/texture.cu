#include "hip/hip_runtime.h"
#include <cmath>
#include "texture.h"
#include "mesh.cuh"

#include "intellisense_cuda.h"

inline void CudaCheck(hipError_t status)
{
	if ( status != hipSuccess )
	{
		throw std::runtime_error("CUDA Error! Error code: " + std::to_string(status));
	}
}

__global__ void textureDownsampling(
	hipSurfaceObject_t texOut,
	hipTextureObject_t texIn,
	uint32_t width,
	uint32_t height
) {
	uint32_t pixelX = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t pixelY = threadIdx.y + blockIdx.y * blockDim.y;
	if (pixelX >= width || pixelY >= height) return;

	float inv_w = 1.0f / float(width);
	float inv_h = 1.0f / float(height);

	uchar4 rgba = tex2D<uchar4>(texIn, inv_w * (0.5f + pixelX) , inv_h * (0.5f + pixelY));

	surf2Dwrite(rgba, texOut, pixelX, pixelY);
}

Texture::Texture(const std::string& file) : Picture(file)
{
	if ( m_channels < 4 )
	{
		throw std::runtime_error("Failed to create texture! Choose picture with 4 channels.");
	}

	auto basePtr = m_data.data();
	m_numLevels = static_cast<uint32_t>(std::floor(std::log2(std::max(m_width, m_height))));

	hipExtent extent = make_hipExtent(m_width, m_height, 0);
	m_cuChannelDesc = hipCreateChannelDesc<uchar4>();

	CudaCheck(hipMallocMipmappedArray(&m_cuMipmapArray, &m_cuChannelDesc, extent, m_numLevels));

	hipArray_t baseLevel;
	CudaCheck(hipGetMipmappedArrayLevel(&baseLevel, m_cuMipmapArray, 0));

	hipMemcpy3DParms copyParams{};
	copyParams.srcPtr = make_hipPitchedPtr(baseLevel, m_pitch, m_width, m_height);
	copyParams.dstArray = baseLevel;
	copyParams.extent.width = m_width;
	copyParams.extent.height = m_height;
	copyParams.extent.depth = 1;
	copyParams.kind = hipMemcpyHostToDevice;
	CudaCheck(hipMemcpy3D(&copyParams));

	// generate multilevel mipmap
	for ( uint32_t level = 1; level < m_numLevels; ++level )
	{
		hipArray_t levelFrom;
		hipArray_t levelTo;

		CudaCheck(hipGetMipmappedArrayLevel(&levelTo, m_cuMipmapArray, level));
		CudaCheck(hipGetMipmappedArrayLevel(&levelFrom, m_cuMipmapArray, level - 1));

		hipExtent extentLevelTo{};
		CudaCheck(hipArrayGetInfo(nullptr, &extentLevelTo, nullptr, levelTo));

		hipExtent extentLevelFrom{};
		CudaCheck(hipArrayGetInfo(nullptr, &extentLevelFrom, nullptr, levelFrom));

		uint32_t width = m_width >> level;
		uint32_t height = m_height >> level;

		hipTextureObject_t texIn;

		hipResourceDesc texResDesc{};
		memset(&texResDesc, 0, sizeof(hipResourceDesc));
		texResDesc.resType = hipResourceTypeArray;
		texResDesc.res.array.array = levelFrom;

		hipTextureDesc texDesc{};
		texDesc.normalizedCoords = 1;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.addressMode[2] = hipAddressModeClamp;
		texDesc.readMode = hipReadModeElementType;

		CudaCheck(hipCreateTextureObject(&texIn, &texResDesc, &texDesc, nullptr));

		hipSurfaceObject_t surfaceOut;
		hipResourceDesc surfDesc{};
		memset(&surfDesc, 0, sizeof(hipResourceDesc));
		surfDesc.resType = hipResourceTypeArray;
		surfDesc.res.array.array = levelTo;
		CudaCheck(hipCreateSurfaceObject(&surfaceOut, &surfDesc));

		dim3 blkDim(16, 16);
		dim3 gridDim((width + blkDim.x - 1) / blkDim.x, (height + blkDim.y - 1) / blkDim.y);
		textureDownsampling KERNEL_DIM(gridDim, blkDim) (surfaceOut, texIn, width, height);

		CudaCheck(hipDeviceSynchronize());
		CudaCheck(hipDestroyTextureObject(texIn));
		CudaCheck(hipDestroySurfaceObject(surfaceOut));
	}

	// create mipmap texture
	hipResourceDesc mipmapTexResDes{};
	mipmapTexResDes.resType = hipResourceTypeMipmappedArray;
	mipmapTexResDes.res.mipmap.mipmap = m_cuMipmapArray;

	memset(&m_cuTextureDesc, 0, sizeof(hipTextureDesc));
	m_cuTextureDesc.normalizedCoords = 1;
	m_cuTextureDesc.filterMode = hipFilterModeLinear;
	m_cuTextureDesc.addressMode[0] = hipAddressModeClamp;
	m_cuTextureDesc.addressMode[1] = hipAddressModeClamp;
	m_cuTextureDesc.addressMode[2] = hipAddressModeClamp;
	m_cuTextureDesc.maxMipmapLevelClamp = m_numLevels - 1;
	m_cuTextureDesc.readMode = hipReadModeElementType;

	CudaCheck(hipCreateTextureObject(&m_cuTextureObj, &mipmapTexResDes, &m_cuTextureDesc, nullptr));
}

Texture::~Texture()
{
	if ( m_cuMipmapArray )
	{
		CudaCheck(hipFreeMipmappedArray(m_cuMipmapArray));
	}
	if ( m_cuTextureObj )
	{
		CudaCheck(hipDestroyTextureObject(m_cuTextureObj));
	}
}
